#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

__global__ void boxblur( unsigned char * rgb, unsigned char * rgb2, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i>2 && i < cols-2 && j>1 && j < rows ) {
  
  for(int k=0; k<3; ++k){
      auto res =     rgb[ 3 * ((j-1) * cols + i-1) + k ]    + rgb[ 3 * ((j-1) * cols + i) + k ]     + rgb[ 3 * ((j-1) * cols + i+1) + k ]   +
                     rgb[ 3 * (j * cols + i-1) + k ]        + rgb[ 3 * (j * cols + i) + k ]         + rgb[ 3 * (j * cols + i+1) + k]        +
                     rgb[ 3 * ((j+1) * cols + i-1) + k ]    + rgb[ 3 * ((j+1) * cols + i) + k]      + rgb[ 3 * ((j+1) * cols + i+1) + k ];
                               
      rgb2[ 3 *(j * cols + i) + k ] = res/9;
  }
  
    
  }  
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  //auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols; 

  // Copie de l'image en entrée dans une mémoire dite "pinned" de manière à accélérer les transferts.
  // OpenCV alloue la mémoire en interne lors de la décompression de l'image donc soit sans doute avec
  // un malloc standard.
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );
  
  std::memcpy( rgb, m_in.data, 3 * rows * cols );
  
  unsigned char * rgb2 = nullptr;
  hipHostMalloc( &rgb2, 3 * rows * cols );
  
  cv::Mat m_out( rows, cols, CV_8UC3, rgb2 );

  unsigned char * rgb_d;
  unsigned char * rgb2_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &rgb2_d, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  hipMemcpy( rgb2_d, rgb2, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 32, 16 );
  dim3 grid0( ( cols  - 1) / block.x + 1 , ( rows - 1 ) / block.y + 1 );
    
  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel uniquement.
  hipEventRecord( start );

  
  boxblur<<< grid0, block >>>( rgb_d, rgb2_d, cols, rows * 3 );

  hipEventRecord( stop );
  
  hipMemcpy( rgb2, rgb2_d, 3 * rows * cols, hipMemcpyDeviceToHost );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "outBoxblur_cu.jpg", m_out );

  hipFree( rgb_d);
  hipFree( rgb2_d);

  hipHostFree( rgb );
  hipHostFree( rgb2 );
  
  return 0;
}
