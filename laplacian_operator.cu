#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

/**
 * Kernel pour transformer l'image RGB en niveaux de gris.
 */
__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }
}

/**
 * Kernel pour obtenir les contours à partir de l'image en niveaux de gris.
 */
__global__ void laplacian_operator( unsigned char * g, unsigned char * s, std::size_t cols, std::size_t rows )
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if( i > 1 && i < cols && j > 1 && j < rows )
  {
    auto res = - g[ (j-1)*cols + i - 1 ] - g[ (j-1)*cols + i ]   - g[ (j-1)*cols + i + 1 ]
               - g[ (j)*cols + i - 1 ]   + 8 * g[ (j)*cols + i ] - g[ (j)*cols + i + 1 ] 
               - g[ (j+1)*cols + i - 1 ] - g[ (j+1)*cols + i ]   - g[ (j+1)*cols + i + 1 ];

    res = res > 128 ? res : 0;
    s[ j * cols + i ] = res;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  //auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  //std::vector< unsigned char > g( rows * cols );
  // Allocation de l'image de sortie en RAM côté CPU.
  unsigned char * g = nullptr;
  hipHostMalloc( &g, rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g );

  // Copie de l'image en entrée dans une mémoire dite "pinned" de manière à accélérer les transferts.
  // OpenCV alloue la mémoire en interne lors de la décompression de l'image donc soit sans doute avec
  // un malloc standard.
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );
  
  std::memcpy( rgb, m_in.data, 3 * rows * cols );

  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * s_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 32, 16 );
  dim3 grid0( ( cols - 1) / block.x + 1 , ( rows - 1 ) / block.y + 1 );
    
  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel uniquement.
  grayscale<<< grid0, block >>>( rgb_d, g_d, cols, rows );
  hipEventRecord( start );

  laplacian_operator<<< grid0, block >>>( g_d, s_d, cols, rows );  

  
  hipEventRecord( stop );
  
  hipMemcpy( g, s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "outLaplacian_cu.jpg", m_out );

  hipFree( rgb_d);
  hipFree( g_d);
  hipFree( s_d);

  hipHostFree( g );
  hipHostFree( rgb );
  
  return 0;
}
